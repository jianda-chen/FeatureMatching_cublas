#include "cuda_match.h"
#include <cstdio>
#include <vector>

#include <hipblas.h>
#include <hip/hip_math_constants.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>



#define BATCH_SIZE 128

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        assert(0);
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static const char *_cublasGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";

	case HIPBLAS_STATUS_NOT_SUPPORTED:
		return "HIPBLAS_STATUS_NOT_SUPPORTED";

	case HIPBLAS_STATUS_UNKNOWN:
		return "HIPBLAS_STATUS_UNKNOWN";
	}

	return "unknown CUBLAS error";
}

inline void __cublasSafeCall(hipblasStatus_t err, const char *file, const int line)
{
	if (HIPBLAS_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUBLAS error in file '%s', line %d \n error  %s\nterminating!\n", file, line, 
			_cublasGetErrorEnum(err)); 
			hipDeviceReset(); assert(0); 
	}
}


inline int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }
#define cublasSafeCall( err)  __cublasSafeCall(err, __FILE__, __LINE__)


int cuda_match_two_set(const float *A, const int K1, const float *B, const int K2, const int Ndims, float *C, const float Thresh)
{    
	
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

   

    thrust::device_vector<float> x(BATCH_SIZE * iDivUp(K1, BATCH_SIZE) * Ndims);
    thrust::copy(A, A + K1 * Ndims, x.begin());
    thrust::device_vector<float> y(B, B + K2 * Ndims);    
   

    thrust::host_vector<float> h_dots(K2 * BATCH_SIZE);
    thrust::device_vector<float> d_dots = h_dots;

    thrust::host_vector<float> h_dist(K2 * BATCH_SIZE);
    thrust::device_vector<float> d_dist(K2 * BATCH_SIZE);

    

	thrust::host_vector<int> cpu_min_index(BATCH_SIZE * iDivUp(K1, BATCH_SIZE));
	thrust::host_vector<int> cpu_min_index2(BATCH_SIZE * iDivUp(K1, BATCH_SIZE));
	thrust::host_vector<float> cpu_min_value2(BATCH_SIZE * iDivUp(K1, BATCH_SIZE));
	thrust::host_vector<float> cpu_min_value(BATCH_SIZE * iDivUp(K1, BATCH_SIZE));

	
    float alpha = 1.f;
    float beta  = 0.f;
   
    for (int b = 0; b < iDivUp(K1, BATCH_SIZE); b++)
    {
    	alpha = 1.f;
    	cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, K2, BATCH_SIZE, Ndims, &alpha,
                               thrust::raw_pointer_cast(y.data()), Ndims, 
                               (float*)(thrust::raw_pointer_cast(x.data())) + b * BATCH_SIZE * Ndims , Ndims, &beta,
                               thrust::raw_pointer_cast(d_dots.data()), K2));
    	
#ifdef DEBUG
    	printf("(A*B)\n");
    	h_dots = d_dots;
    	for(int i = 0; i < BATCH_SIZE; i++)
    	{
		    for(int j = 0; j < K2; j++)
		    	printf("%f ", h_dots[i*K2+j]);
		    printf("\n");
    	}
    	fflush( stdout );
#endif

    	alpha = -2.f;
    	thrust::fill(d_dist.begin(), d_dist.end(), 2.f);
    	cublasSafeCall(hipblasSaxpy(handle, K2 * BATCH_SIZE, &alpha, 
    								thrust::raw_pointer_cast(d_dots.data()), 1,
    								thrust::raw_pointer_cast(d_dist.data()), 1));
#ifdef DEBUG
    	printf("2 - 2(A*B)\n");
    	h_dist = d_dist;
    	for(int i = 0; i < BATCH_SIZE; i++)
    	{
		    for(int j = 0; j < K2; j++)
		    	printf("%f ", h_dist[i*K2+j]);
		    printf("\n");
    	}
    	fflush( stdout );
#endif
    	//find best and second best
    	for (int i = 0; i < BATCH_SIZE; i++)
		{
			hipblasIsamin(handle, K2, ((float*)thrust::raw_pointer_cast(d_dist.data())) + K2 * i, 1, 
						((int*)thrust::raw_pointer_cast(cpu_min_index.data())) + i + b * BATCH_SIZE);
		}		
		h_dist = d_dist;
		for(int i  = 0; i < BATCH_SIZE; i++)
		{
			cpu_min_value[i + b * BATCH_SIZE] = h_dist[i*K2+cpu_min_index[i + b * BATCH_SIZE] - 1];
			h_dist[i*K2+cpu_min_index[i + b * BATCH_SIZE] -1] = 100000.f;
		}
		d_dist = h_dist;	    
		for (int i = 0; i < BATCH_SIZE; i++)
		{
			hipblasIsamin(handle, K2, ((float*)thrust::raw_pointer_cast(d_dist.data())) + K2 * i, 1, 
						((int*)thrust::raw_pointer_cast(cpu_min_index2.data())) + i + b * BATCH_SIZE);
		}
		for(int i  = 0; i < BATCH_SIZE; i++)
		{
			cpu_min_value2[i + b * BATCH_SIZE] = h_dist[i*K2+cpu_min_index2[i + b * BATCH_SIZE] - 1];
		}
    }

#ifdef DEBUG
	for (int i = 0; i < K1; i++)
	{
		printf("%d ", cpu_min_index[i]);
	}
	for (int i = 0; i < K1; i++)
	{
		printf("%d ", cpu_min_index2[i]);
	}
#endif

	int num_match = 0;
	for (int i = 0; i < K1; i++)
	{
		if (cpu_min_value[i] * Thresh < cpu_min_value2[i])
		{
			C[num_match*2] = i + 1; // for MATLAB 1-based index
			C[num_match*2 + 1] = cpu_min_index[i];
			num_match++;
		}
	}

	cublasSafeCall(hipblasDestroy(handle));
	return num_match;
}




